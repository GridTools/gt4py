#include "hip/hip_runtime.h"
#include "atlas/grid.h"
#include "atlas/mesh/actions/BuildCellCentres.h"
#include "atlas/mesh/actions/BuildDualMesh.h"
#include "atlas/mesh/actions/BuildEdges.h"
#include "atlas/meshgenerator.h"
#include "gridtools/common/integral_constant.hpp"
#include <array_fwd.h>
#include <atlas/array.h>
#include <atlas/grid/StructuredGrid.h>
#include <atlas/mesh.h>
#include <atlas/option.h>
#include <field/Field.h>
#include <functionspace/EdgeColumns.h>
#include <type_traits>

#include "gridtools/next/atlas_array_view_adapter.hpp"
#include <gridtools/next/atlas_adapter.hpp>
#include <gridtools/next/atlas_field_util.hpp>
#include <gridtools/next/mesh.hpp>
#include <gridtools/sid/synthetic.hpp>

#include "tests/include/util/atlas_util.hpp"

namespace dim {
    struct k;
} // namespace dim

template <class Ptr, class Strides, class UpperBounds>
__global__ void kernel(Ptr ptr_holder, Strides strides, UpperBounds upper_bounds) {
    auto ptr = ptr_holder();
    gridtools::sid::shift(ptr, gridtools::device::at_key<edge>(strides), threadIdx.x);
    for (int i = 0; i < gridtools::device::at_key<dim::k>(upper_bounds); ++i) {
        printf("%f\n", *ptr);
        gridtools::sid::shift(ptr, gridtools::device::at_key<dim::k>(strides), 1);
    }
}

int main() {
    auto mesh = atlas_util::make_mesh();
    atlas::mesh::actions::build_edges(mesh);

    int nb_levels = 5;
    atlas::functionspace::EdgeColumns fs_edges(mesh, atlas::option::levels(nb_levels) | atlas::option::halo(1));

    atlas::Field f;
    auto my_field = fs_edges.createField<double>(atlas::option::name("my_field"));

    auto view = atlas::array::make_view<double, 2>(my_field);
    for (int i = 0; i < fs_edges.size(); ++i)
        for (int k = 0; k < nb_levels; ++k)
            view(i, k) = i * 10 + k;

    auto my_field_as_data_store =
        gridtools::next::atlas_util::as_data_store<edge, dim::k>::with_type<double>{}(my_field);
    static_assert(gridtools::is_sid<decltype(my_field_as_data_store)>{});

    kernel<<<1, fs_edges.size()>>>(gridtools::sid::get_origin(my_field_as_data_store),
        gridtools::sid::get_strides(my_field_as_data_store),
        gridtools::sid::get_upper_bounds(my_field_as_data_store));
    hipDeviceSynchronize();
}
