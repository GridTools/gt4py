#include "hip/hip_runtime.h"
/*
 * GridTools
 *
 * Copyright (c) 2014-2021, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include "./test_hypercube_iterator.cpp"
#include <gridtools/common/cuda_util.hpp>

static const size_t Size = 2;

GT_FUNCTION int linear_index(gridtools::array<size_t, 2> &index) { return index[0] * Size + index[1]; }

__global__ void test_kernel(int *out_ptr) {
    for (size_t i = 0; i < Size * Size; ++i)
        out_ptr[i] = -1;

    using hypercube_t = gridtools::array<gridtools::array<size_t, 2>, 2>;
    for (auto pos : make_hypercube_view(hypercube_t{{{0ul, Size}, {0ul, Size}}})) {
        out_ptr[linear_index(pos)] = linear_index(pos);
    }
};

TEST(multi_iterator, iterate_on_device) {
    int *out;
    GT_CUDA_CHECK(hipMalloc(&out, sizeof(int) * Size * Size));

    test_kernel<<<1, 1>>>(out);

    int host_out[Size * Size];
    GT_CUDA_CHECK(hipMemcpy(&host_out, out, sizeof(int) * Size * Size, hipMemcpyDeviceToHost));

    for (size_t i = 0; i < Size * Size; ++i)
        ASSERT_EQ(i, host_out[i]) << "at i = " << i;
}
