#include "hip/hip_runtime.h"
/*
 * GridTools
 *
 * Copyright (c) 2014-2021, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */
#include "test_gt_math.cpp"

#include <cuda_test_helper.hpp>

TEST(math_cuda, test_fabs) { EXPECT_TRUE(on_device::exec(GT_MAKE_INTEGRAL_CONSTANT_FROM_VALUE(&test_fabs))); }
TEST(math_cuda, test_abs) { EXPECT_TRUE(on_device::exec(GT_MAKE_INTEGRAL_CONSTANT_FROM_VALUE(&test_fabs))); }

TEST(math_cuda, test_log) {
    EXPECT_TRUE(on_device::exec(GT_MAKE_INTEGRAL_CONSTANT_FROM_VALUE(&test_log<double>), 2.3, std::log(2.3)));
    EXPECT_TRUE(on_device::exec(GT_MAKE_INTEGRAL_CONSTANT_FROM_VALUE(&test_log<float>), 2.3f, std::log(2.3f)));
}

TEST(math_cuda, test_exp) {
    EXPECT_TRUE(on_device::exec(GT_MAKE_INTEGRAL_CONSTANT_FROM_VALUE(&test_exp<double>), 2.3, std::exp(2.3)));
    EXPECT_TRUE(on_device::exec(GT_MAKE_INTEGRAL_CONSTANT_FROM_VALUE(&test_exp<float>), 2.3f, std::exp(2.3f)));
}

TEST(math_cuda, test_pow) {
    EXPECT_TRUE(on_device::exec(GT_MAKE_INTEGRAL_CONSTANT_FROM_VALUE(&test_pow<double>), 2.3, std::pow(2.3, 2.3)));
    EXPECT_TRUE(on_device::exec(GT_MAKE_INTEGRAL_CONSTANT_FROM_VALUE(&test_pow<float>), 2.3f, std::pow(2.3f, 2.3f)));
}
