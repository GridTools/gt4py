/*
 * GridTools
 *
 * Copyright (c) 2014-2021, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */
#include <gridtools/common/array.hpp>
#include <gridtools/fn/unstructured.hpp>

#include <gtest/gtest.h>

#include <gridtools/fn/backend/gpu.hpp>
#include <gridtools/sid/synthetic.hpp>

namespace gridtools::fn {
    namespace {
        using namespace literals;
        using sid::property;

        template <int I>
        using int_t = integral_constant<int, I>;

        template <class C, int MaxNeighbors>
        struct stencil {
            GT_FUNCTION constexpr auto operator()() const {
                return [](auto const &in) {
                    int tmp = 0;
                    tuple_util::host_device::for_each(
                        [&](auto i) {
                            auto shifted = shift(in, C(), i);
                            if (can_deref(shifted))
                                tmp += deref(shifted);
                        },
                        meta::rename<tuple, meta::make_indices_c<MaxNeighbors>>());
                    return tmp;
                };
            }
        };

        struct v2v {};
        struct v2e {};

        using block_sizes_t = meta::list<meta::list<unstructured::dim::horizontal, int_t<32>>,
            meta::list<unstructured::dim::vertical, int_t<1>>>;

        TEST(unstructured, v2v_sum) {
            auto apply_stencil = [](auto executor, auto &out, auto const &in) {
                executor().arg(out).arg(in).assign(0_c, stencil<v2v, 3>(), 1_c).execute();
            };
            auto fencil = [&](auto const &v2v_table, int nvertices, int nlevels, auto &out, auto const &in) {
                auto v2v_conn = connectivity<v2v>(v2v_table);
                auto domain = unstructured_domain({nvertices, nlevels}, {}, v2v_conn);
                auto backend = make_backend(backend::gpu<block_sizes_t>(), domain);
                apply_stencil(backend.stencil_executor(), out, in);
            };

            auto v2v_table = cuda_util::cuda_malloc<array<int, 3>>(3);
            int v2v_tableh[3][3] = {{1, 2, -1}, {0, 2, -1}, {0, 1, -1}};
            hipMemcpy(v2v_table.get(), v2v_tableh, 3 * sizeof(array<int, 3>), hipMemcpyHostToDevice);

            auto in = cuda_util::cuda_malloc<int>(3 * 5);
            auto out = cuda_util::cuda_malloc<int>(3 * 5);
            int inh[3][5], outh[3][5] = {};
            for (int v = 0; v < 3; ++v)
                for (int k = 0; k < 5; ++k)
                    inh[v][k] = 5 * v + k;
            hipMemcpy(in.get(), inh, 3 * 5 * sizeof(int), hipMemcpyHostToDevice);

            auto as_synthetic = [](int *x) {
                return sid::synthetic()
                    .set<property::origin>(sid::host_device::simple_ptr_holder(x))
                    .set<property::strides>(
                        hymap::keys<unstructured::dim::horizontal, unstructured::dim::vertical>::make_values(5_c, 1_c));
            };
            auto in_s = as_synthetic(in.get());
            auto out_s = as_synthetic(out.get());

            GT_CUDA_CHECK(hipDeviceSynchronize());
            fencil(v2v_table.get(), 3, 5, out_s, in_s);
            GT_CUDA_CHECK(hipDeviceSynchronize());
            hipMemcpy(outh, out.get(), 3 * 5 * sizeof(int), hipMemcpyDeviceToHost);

            for (int v = 0; v < 3; ++v)
                for (int k = 0; k < 5; ++k) {
                    int nbsum = 0;
                    for (int i = 0; i < 3; ++i) {
                        int nb = v2v_tableh[v][i];
                        if (nb != -1)
                            nbsum += inh[nb][k];
                    }
                    EXPECT_EQ(outh[v][k], nbsum);
                }
        }

        TEST(unstructured, v2e_sum) {
            auto apply_stencil = [](auto executor, auto &out, auto const &in) {
                executor().arg(out).arg(in).assign(0_c, stencil<v2e, 2>(), 1_c).execute();
            };
            auto fencil = [&](auto const &v2e_table, int nvertices, int nlevels, auto &out, auto const &in) {
                auto v2e_conn = connectivity<v2e>(v2e_table);
                auto domain = unstructured_domain({nvertices, nlevels}, {}, v2e_conn);
                auto backend = make_backend(backend::gpu<block_sizes_t>(), domain);
                apply_stencil(backend.stencil_executor(), out, in);
            };

            auto v2e_table = cuda_util::cuda_malloc<array<int, 2>>(3);
            int v2e_tableh[3][2] = {{0, 2}, {0, 1}, {1, 2}};
            hipMemcpy(v2e_table.get(), v2e_tableh, 3 * sizeof(array<int, 2>), hipMemcpyHostToDevice);

            auto in = cuda_util::cuda_malloc<int>(3 * 5);
            auto out = cuda_util::cuda_malloc<int>(3 * 5);
            int inh[3][5], outh[3][5] = {};
            for (int e = 0; e < 3; ++e)
                for (int k = 0; k < 5; ++k)
                    inh[e][k] = 5 * e + k;
            hipMemcpy(in.get(), inh, 3 * 5 * sizeof(int), hipMemcpyHostToDevice);

            auto as_synthetic = [](int *x) {
                return sid::synthetic()
                    .set<property::origin>(sid::host_device::simple_ptr_holder(x))
                    .set<property::strides>(
                        hymap::keys<unstructured::dim::horizontal, unstructured::dim::vertical>::make_values(5_c, 1_c));
            };
            auto in_s = as_synthetic(in.get());
            auto out_s = as_synthetic(out.get());

            GT_CUDA_CHECK(hipDeviceSynchronize());
            fencil(v2e_table.get(), 3, 5, out_s, in_s);
            GT_CUDA_CHECK(hipDeviceSynchronize());
            hipMemcpy(outh, out.get(), 3 * 5 * sizeof(int), hipMemcpyDeviceToHost);

            for (int v = 0; v < 3; ++v)
                for (int k = 0; k < 5; ++k) {
                    int nbsum = 0;
                    for (int i = 0; i < 2; ++i) {
                        int nb = v2e_tableh[v][i];
                        nbsum += inh[nb][k];
                    }
                    EXPECT_EQ(outh[v][k], nbsum);
                }
        }

    } // namespace
} // namespace gridtools::fn
