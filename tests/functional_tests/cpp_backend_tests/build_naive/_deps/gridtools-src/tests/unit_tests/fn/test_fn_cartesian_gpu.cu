/*
 * GridTools
 *
 * Copyright (c) 2014-2021, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */
#include <gridtools/fn/cartesian.hpp>

#include <gtest/gtest.h>

#include <gridtools/fn/backend/gpu.hpp>

namespace gridtools::fn {
    namespace {
        using namespace literals;
        using sid::property;

        template <int I>
        using int_t = integral_constant<int, I>;

        struct stencil {
            constexpr auto operator()() const {
                using namespace cartesian::dim;
                return [](auto const &in) { return deref(shift(in, i(), 1_c)); };
            }
        };

        struct fwd_sum_scan : fwd {
            static GT_FUNCTION constexpr auto body() {
                return scan_pass(
                    [](auto acc, auto const &iter) { return acc + deref(iter); }, [](auto acc) { return acc; });
            }
        };

        struct bwd_sum_scan : bwd {
            static GT_FUNCTION constexpr auto body() {
                return scan_pass(
                    [](auto acc, auto const &iter) { return acc + deref(iter); }, [](auto acc) { return acc; });
            }
        };

        TEST(cartesian, stencil) {
            using block_sizes_t = meta::list<meta::list<cartesian::dim::i, int_t<32>>,
                meta::list<cartesian::dim::j, int_t<8>>,
                meta::list<cartesian::dim::k, int_t<1>>>;
            auto apply_stencil = [](auto executor, auto &out, auto const &in) {
                executor().arg(out).arg(in).assign(0_c, stencil(), 1_c).execute();
            };

            auto fencil = [&](auto const &sizes, auto &out, auto const &in) {
                auto be = backend::gpu<block_sizes_t>();
                auto alloc = tmp_allocator(be);
                auto tmp = allocate_global_tmp<int>(alloc, sizes);
                auto domain = cartesian_domain(std::array<int, 3>{sizes[0] - 1, sizes[1], sizes[2]});
                auto backend = make_backend(be, domain);
                apply_stencil(backend.stencil_executor(), tmp, in);
                apply_stencil(backend.stencil_executor(), out, tmp);
            };

            auto in = cuda_util::cuda_malloc<int>(5 * 3 * 2);
            auto out = cuda_util::cuda_malloc<int>(5 * 3 * 2);
            int inh[5][3][2], outh[5][3][2] = {};
            for (int i = 0; i < 5; ++i)
                for (int j = 0; j < 3; ++j)
                    for (int k = 0; k < 2; ++k)
                        inh[i][j][k] = 6 * i + 2 * j + k;
            hipMemcpy(in.get(), inh, 5 * 3 * 2 * sizeof(int), hipMemcpyHostToDevice);
            auto as_synthetic = [](int *x) {
                return sid::synthetic()
                    .set<property::origin>(sid::host_device::simple_ptr_holder(x))
                    .set<property::strides>(tuple(6_c, 2_c, 1_c));
            };

            auto out_s = as_synthetic(out.get());
            auto in_s = as_synthetic(in.get());
            fencil(std::array{5, 3, 2}, out_s, in_s);

            hipMemcpy(outh, out.get(), 5 * 3 * 2 * sizeof(int), hipMemcpyDeviceToHost);

            for (int i = 0; i < 3; ++i)
                for (int j = 0; j < 3; ++j)
                    for (int k = 0; k < 2; ++k)
                        EXPECT_EQ(outh[i][j][k], 6 * (i + 2) + 2 * j + k);
        }

        TEST(cartesian, vertical) {
            using block_sizes_t = meta::list<meta::list<cartesian::dim::i, int_t<32>>,
                meta::list<cartesian::dim::j, int_t<8>>,
                meta::list<cartesian::dim::k, int_t<1>>>;
            auto apply_double_scan = [](auto executor, auto &a, auto &b, auto const &c) {
                executor()
                    .arg(a)
                    .arg(b)
                    .arg(c)
                    .assign(1_c, fwd_sum_scan(), 42, 2_c)
                    .assign(0_c, bwd_sum_scan(), 8, 1_c)
                    .execute();
            };

            auto double_scan = [&](auto sizes, auto &a, auto &b, auto const &c) {
                auto domain = cartesian_domain(sizes);
                auto backend = make_backend(backend::gpu<block_sizes_t>(), domain);
                apply_double_scan(backend.vertical_executor(), a, b, c);
            };

            std::array<int, 3> sizes = {5, 3, 2};
            auto a = cuda_util::cuda_malloc<int>(5 * 3 * 2);
            auto b = cuda_util::cuda_malloc<int>(5 * 3 * 2);
            auto c = cuda_util::cuda_malloc<int>(5 * 3 * 2);
            int ah[5][3][2] = {}, bh[5][3][2] = {}, ch[5][3][2];
            for (int i = 0; i < 5; ++i)
                for (int j = 0; j < 3; ++j)
                    for (int k = 0; k < 2; ++k)
                        ch[i][j][k] = 6 * i + 2 * j + k;
            hipMemcpy(c.get(), ch, 5 * 3 * 2 * sizeof(int), hipMemcpyHostToDevice);
            auto as_synthetic = [](int *x) {
                return sid::synthetic()
                    .set<property::origin>(sid::host_device::simple_ptr_holder(x))
                    .set<property::strides>(tuple(6_c, 2_c, 1_c));
            };

            auto a_s = as_synthetic(a.get());
            auto b_s = as_synthetic(b.get());
            auto c_s = as_synthetic(c.get());
            double_scan(sizes, a_s, b_s, c_s);
            hipMemcpy(bh, b.get(), 5 * 3 * 2 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(ah, a.get(), 5 * 3 * 2 * sizeof(int), hipMemcpyDeviceToHost);

            for (int i = 0; i < 5; ++i)
                for (int j = 0; j < 3; ++j) {
                    int res = 42;
                    for (int k = 0; k < 2; ++k) {
                        res += ch[i][j][k];
                        EXPECT_EQ(bh[i][j][k], res);
                    }
                    res = 8;
                    for (int k = 1; k >= 0; --k) {
                        res += bh[i][j][k];
                        EXPECT_EQ(ah[i][j][k], res);
                    }
                }
        }
    } // namespace
} // namespace gridtools::fn
