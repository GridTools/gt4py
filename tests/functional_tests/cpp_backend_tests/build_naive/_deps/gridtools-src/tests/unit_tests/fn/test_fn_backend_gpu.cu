/*
 * GridTools
 *
 * Copyright (c) 2014-2021, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */
#include <gridtools/fn/backend/gpu.hpp>

#include <gtest/gtest.h>

#include <gridtools/fn/column_stage.hpp>
#include <gridtools/sid/composite.hpp>
#include <gridtools/sid/synthetic.hpp>

#include <cuda_test_helper.hpp>

namespace gridtools::fn::backend {
    namespace {
        using namespace literals;
        using sid::property;

        template <int I>
        using int_t = integral_constant<int, I>;

        struct sum_scan : fwd {
            static GT_FUNCTION constexpr auto body() {
                return scan_pass(
                    [](auto acc, auto const &iter) { return tuple(get<0>(acc) + *iter, get<1>(acc) * *iter); },
                    [](auto acc) { return get<0>(acc); });
            }
        };

        struct make_iterator_mock {
            GT_FUNCTION auto operator()() const {
                return
                    [](auto tag, auto const &ptr, auto const &strides) { return device::at_key<decltype(tag)>(ptr); };
            }
        };

        TEST(backend_gpu, apply_column_stage) {
            auto in = cuda_util::cuda_malloc<int>(5 * 7 * 3);
            auto out = cuda_util::cuda_malloc<int>(5 * 7 * 3);
            int inh[5][7][3], outh[5][7][3] = {};
            for (int i = 0; i < 5; ++i)
                for (int j = 0; j < 7; ++j)
                    for (int k = 0; k < 3; ++k)
                        inh[i][j][k] = 21 * i + 3 * j + k;
            hipMemcpy(in.get(), inh, 5 * 7 * 3 * sizeof(int), hipMemcpyHostToDevice);

            auto as_synthetic = [](int *x) {
                return sid::synthetic()
                    .set<property::origin>(sid::host_device::simple_ptr_holder(x))
                    .set<property::strides>(tuple(21_c, 3_c, 1_c));
            };

            auto composite =
                sid::composite::keys<int_t<0>, int_t<1>>::make_values(as_synthetic(out.get()), as_synthetic(in.get()));

            auto sizes = hymap::keys<int_t<0>, int_t<1>, int_t<2>>::values<int_t<5>, int_t<7>, int_t<3>>();

            column_stage<int_t<1>, sum_scan, 0, 1> cs;

            using block_sizes_t = meta::list<meta::list<int_t<0>, int_t<4>>, meta::list<int_t<2>, int_t<2>>>;

            apply_column_stage(
                gpu<block_sizes_t>(), sizes, cs, make_iterator_mock(), composite, int_t<1>(), tuple(42, 1));

            hipMemcpy(outh, out.get(), 5 * 7 * 3 * sizeof(int), hipMemcpyDeviceToHost);
            for (int i = 0; i < 5; ++i)
                for (int k = 0; k < 3; ++k) {
                    int res = 42;
                    for (int j = 0; j < 7; ++j) {
                        res += inh[i][j][k];
                        EXPECT_EQ(outh[i][j][k], res);
                    }
                }
        }

        TEST(backend_gpu, apply_column_stage_1d) {
            auto in = cuda_util::cuda_malloc<int>(5);
            auto out = cuda_util::cuda_malloc<int>(5);
            int inh[5], outh[5] = {};
            for (int i = 0; i < 5; ++i)
                inh[i] = i;
            hipMemcpy(in.get(), inh, 5 * sizeof(int), hipMemcpyHostToDevice);

            auto as_synthetic = [](int *x) {
                return sid::synthetic()
                    .set<property::origin>(sid::host_device::simple_ptr_holder(x))
                    .set<property::strides>(tuple(1_c));
            };

            auto composite =
                sid::composite::keys<int_t<0>, int_t<1>>::make_values(as_synthetic(out.get()), as_synthetic(in.get()));

            auto sizes = hymap::keys<int_t<0>>::values<int_t<5>>();

            column_stage<int_t<0>, sum_scan, 0, 1> cs;

            using block_sizes_t = meta::list<meta::list<int_t<0>, int_t<4>>, meta::list<int_t<2>, int_t<2>>>;

            apply_column_stage(
                gpu<block_sizes_t>(), sizes, cs, make_iterator_mock(), composite, int_t<0>(), tuple(42, 1));

            hipMemcpy(outh, out.get(), 5 * sizeof(int), hipMemcpyDeviceToHost);
            int res = 42;
            for (int i = 0; i < 5; ++i) {
                res += inh[i];
                EXPECT_EQ(outh[i], res);
            }
        }

        TEST(backend_gpu, apply_column_stage_5d) {
            auto in = cuda_util::cuda_malloc<int>(5 * 7 * 3 * 2 * 3);
            auto out = cuda_util::cuda_malloc<int>(5 * 7 * 3 * 2 * 3);
            int inh[5][7][3][2][3], outh[5][7][3][2][3] = {};
            for (int i = 0; i < 5; ++i)
                for (int j = 0; j < 7; ++j)
                    for (int k = 0; k < 3; ++k)
                        for (int l = 0; l < 2; ++l)
                            for (int m = 0; m < 3; ++m)
                                inh[i][j][k][l][m] = 126 * i + 18 * j + 6 * k + 3 * l + m;
            hipMemcpy(in.get(), inh, 5 * 7 * 3 * 2 * 3 * sizeof(int), hipMemcpyHostToDevice);

            auto as_synthetic = [](int *x) {
                return sid::synthetic()
                    .set<property::origin>(sid::host_device::simple_ptr_holder(x))
                    .set<property::strides>(tuple(126_c, 18_c, 6_c, 3_c, 1_c));
            };

            auto composite =
                sid::composite::keys<int_t<0>, int_t<1>>::make_values(as_synthetic(out.get()), as_synthetic(in.get()));

            auto sizes = hymap::keys<int_t<0>, int_t<1>, int_t<2>, int_t<3>, int_t<4>>::
                values<int_t<5>, int_t<7>, int_t<3>, int_t<2>, int_t<3>>();

            column_stage<int_t<1>, sum_scan, 0, 1> cs;

            using block_sizes_t = meta::list<meta::list<int_t<0>, int_t<4>>,
                meta::list<int_t<2>, int_t<2>>,
                meta::list<int_t<3>, int_t<2>>,
                meta::list<int_t<4>, int_t<1>>>;

            apply_column_stage(
                gpu<block_sizes_t>(), sizes, cs, make_iterator_mock(), composite, int_t<1>(), tuple(42, 1));

            hipMemcpy(outh, out.get(), 5 * 7 * 3 * 2 * 3 * sizeof(int), hipMemcpyDeviceToHost);
            for (int i = 0; i < 5; ++i)
                for (int k = 0; k < 3; ++k) {
                    for (int l = 0; l < 2; ++l) {
                        for (int m = 0; m < 3; ++m) {
                            int res = 42;
                            for (int j = 0; j < 7; ++j) {
                                res += inh[i][j][k][l][m];
                                EXPECT_EQ(outh[i][j][k][l][m], res);
                            }
                        }
                    }
                }
        }

        struct global_tmp_check_fun {
            template <class PtrHolder, class Strides>
            GT_FUNCTION bool operator()(PtrHolder ptr_holder, Strides strides) const {
                auto ptr = ptr_holder();
                for (int i = 0; i < 5; ++i) {
                    for (int j = 0; j < 7; ++j) {
                        for (int k = 0; k < 3; ++k) {
                            *ptr = 21 * i + 3 * j + k;
                            sid::shift(ptr, sid::get_stride<int_t<2>>(strides), 1_c);
                        }
                        sid::shift(ptr, sid::get_stride<int_t<2>>(strides), -3_c);
                        sid::shift(ptr, sid::get_stride<int_t<1>>(strides), 1_c);
                    }
                    sid::shift(ptr, sid::get_stride<int_t<1>>(strides), -7_c);
                    sid::shift(ptr, sid::get_stride<int_t<0>>(strides), 1_c);
                }
                sid::shift(ptr, sid::get_stride<int_t<0>>(strides), -5_c);
                bool correct = true;
                for (int i = 0; i < 5; ++i) {
                    for (int j = 0; j < 7; ++j) {
                        for (int k = 0; k < 3; ++k) {
                            correct &= *ptr == 21 * i + 3 * j + k;
                            sid::shift(ptr, sid::get_stride<int_t<2>>(strides), 1_c);
                        }
                        sid::shift(ptr, sid::get_stride<int_t<2>>(strides), -3_c);
                        sid::shift(ptr, sid::get_stride<int_t<1>>(strides), 1_c);
                    }
                    sid::shift(ptr, sid::get_stride<int_t<1>>(strides), -7_c);
                    sid::shift(ptr, sid::get_stride<int_t<0>>(strides), 1_c);
                }
                return correct;
            }
        };

        TEST(backend_gpu, global_tmp) {
            using block_sizes_t = meta::list<meta::list<int_t<0>, int_t<4>>, meta::list<int_t<2>, int_t<2>>>;
            auto alloc = tmp_allocator(gpu<block_sizes_t>());
            auto sizes = hymap::keys<int_t<0>, int_t<1>, int_t<2>>::values<int_t<5>, int_t<7>, int_t<3>>();
            auto tmp = allocate_global_tmp(alloc, sizes, data_type<int>());
            static_assert(sid::is_sid<decltype(tmp)>());
            auto ptr_holder = sid::get_origin(tmp);
            auto strides = sid::get_strides(tmp);
            bool success = on_device::exec(global_tmp_check_fun(), ptr_holder, strides);
            EXPECT_TRUE(success);
        }
    } // namespace
} // namespace gridtools::fn::backend
