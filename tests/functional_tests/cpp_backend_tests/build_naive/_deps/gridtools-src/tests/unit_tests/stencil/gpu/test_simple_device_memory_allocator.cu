#include "hip/hip_runtime.h"
/*
 * GridTools
 *
 * Copyright (c) 2014-2021, ETH Zurich
 * All rights reserved.
 *
 * Please, refer to the LICENSE file in the root directory.
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <gtest/gtest.h>

#include <gridtools/common/cuda_util.hpp>
#include <gridtools/common/integral_constant.hpp>
#include <gridtools/meta.hpp>
#include <gridtools/sid/allocator.hpp>

#include <cuda_test_helper.hpp>

namespace gridtools {
    namespace {

        template <typename PtrHolder>
        __device__ bool check_allocation(PtrHolder ptr_holder) {
            auto &ref = *ptr_holder();
            ref = 1.;
            return ref == 1.;
        }

        template <typename PtrHolder>
        __global__ void test_allocated(PtrHolder testee, bool *result) {}

        TEST(simple_device_memory_allocator, test) {
            sid::device::allocator<GT_INTEGRAL_CONSTANT_FROM_VALUE(&cuda_util::cuda_malloc<char[]>)> alloc;
            auto ptr_holder = allocate(alloc, meta::lazy::id<double>{}, 1);

            auto result = gridtools::on_device::exec(
                GT_MAKE_INTEGRAL_CONSTANT_FROM_VALUE(&check_allocation<decltype(ptr_holder)>), ptr_holder);
            ASSERT_TRUE(result);
        }
    } // namespace
} // namespace gridtools
